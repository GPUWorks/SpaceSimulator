#include "hip/hip_runtime.h"
#include "../include/NbodySystem.cuh"
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

// Vector of vertes resources to all bodies
//static std::vector<struct hipGraphicsResource*> *resources = new std::vector<struct hipGraphicsResource*>();
static std::vector<struct hipGraphicsResource*> resources;

// Kernel prototypes
void __global__ moveBodyKernel(double3 *vertices, int num_vertices);

// CUDA Memory
__constant__ double cuda_translation_matrix[16];

void initializeNbodySystem(Config *config){
	if((config->getDebugLevel() & 0x10) == 16){	
		printf("NbodySystem.cu\t\tInitializing\n");	
	}
}

void addBodyVertexBuffer(GLuint buffer, Config *config){
	if((config->getDebugLevel() & 0x0) == 8){		
		printf("NbodySystem.cu\t\tAdding vertex buffer %d\n", buffer);
	}
	
	// Creating new cuda resource
	struct hipGraphicsResource *resource;
	hipGraphicsGLRegisterBuffer(&resource, buffer, hipGraphicsRegisterFlagsNone);
	
	// Adding the new resource
	resources.push_back(resource);
}

void moveBody(int bodyIndex, int numVertices, double *translation){
	// Local vars
	double3 *vertices = 0;
	size_t num_bytes_vertices;
	
	// Getting the vertex array pointer
	hipGraphicsMapResources(1, &resources[bodyIndex]);
	hipGraphicsResourceGetMappedPointer((void**)&vertices, &num_bytes_vertices, resources[bodyIndex]);
	
	// Transferring the translation matrix
	hipMemcpyToSymbol(HIP_SYMBOL(cuda_translation_matrix), translation, 4*4*sizeof(double), 0, hipMemcpyHostToDevice);
	
	// CUDA call
	dim3 block((numVertices/512) + 1);
	dim3 grid(512);
	moveBodyKernel<<<block, grid>>>(vertices, numVertices);
	
	// Unmapping, making ready for rendering
	hipGraphicsUnmapResources(1, &resources[bodyIndex]);
	
	// Error check
	hipError_t error = hipGetLastError();
	if(error != 0){	
		printf("NbodySystem.cu\t\tError: %s\n", hipGetErrorString(error));
	}
}

void __global__ moveBodyKernel(double3 *vertices, int num_vertices){
	// Global index
	int i = (blockIdx.x*blockDim.x) + threadIdx.x;
	
	// Boundary check
	if(i >= num_vertices)return;

	// Multiplying vertex and translation matrix, (Rotation around inclination axis and movement of whole body)
	double vx = cuda_translation_matrix[0]*vertices[i].x + cuda_translation_matrix[4]*vertices[i].y + cuda_translation_matrix[8]*vertices[i].z + cuda_translation_matrix[12];
	double vy = cuda_translation_matrix[1]*vertices[i].x + cuda_translation_matrix[5]*vertices[i].y + cuda_translation_matrix[9]*vertices[i].z + cuda_translation_matrix[13];
	double vz = cuda_translation_matrix[2]*vertices[i].x + cuda_translation_matrix[6]*vertices[i].y + cuda_translation_matrix[10]*vertices[i].z + cuda_translation_matrix[14];
	
	// Forcing sphere structure
	vertices[i].x = vx;
	vertices[i].y = vy;
	vertices[i].z = vz;
}
